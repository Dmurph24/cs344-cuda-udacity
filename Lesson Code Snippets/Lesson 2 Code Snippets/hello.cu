#include <stdio.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <hip/device_functions.h>

#define NUM_BLOCKS 4
#define BLOCK_WIDTH 4

__global__ void hello()
{
    printf("Hello world! I'm thread %d in block %d\n", threadIdx.x, blockIdx.x);
}


int main(int argc,char **argv)
{
    // launch the kernel
    hello<<<NUM_BLOCKS, BLOCK_WIDTH>>>();

    // force the printf()s to flush
    hipDeviceSynchronize();

    printf("That's all!\n");

    return 0;
}
