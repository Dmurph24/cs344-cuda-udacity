#include "hip/hip_runtime.h"
#include <stdio.h>
#include <>
#include <hip/device_functions.h>
#include "gputimer.h"
#include "utils.h"

const int BLOCKSIZE	= 128;
const int NUMBLOCKS = 100;					// set this to 1 or 2 for debugging
const int N 		= BLOCKSIZE*NUMBLOCKS;

/* 
 * TODO: modify the foo and bar kernels to use tiling: 
 * 		 - copy the input data to shared memory
 *		 - perform the computation there
 *	     - copy the result back to global memory
 *		 - assume thread blocks of 128 threads
 *		 - handle intra-block boundaries correctly
 * You can ignore boundary conditions (we ignore the first 2 and last 2 elements)
 */
__global__ void foo(float out[], float A[], float B[], float C[], float D[], float E[]){

	int i = threadIdx.x + blockIdx.x*blockDim.x; 
	
	out[i] = (A[i] + B[i] + C[i] + D[i] + E[i]) / 5.0f;
}

__global__ void bar(float out[], float in[]) 
{
	int i = threadIdx.x + blockIdx.x*blockDim.x; 

	out[i] = (in[i-2] + in[i-1] + in[i] + in[i+1] + in[i+2]) / 5.0f;
}

__global__ void bar_tile(float out[], float in[])
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int idx = threadIdx.x;
  extern __shared__ float sh_din[];  
  sh_din[idx + 2] = in[i];
  if (idx == 0) {
    sh_din[idx] = in[i-2];
    sh_din[idx+1] = in[i-1];
  }
  else if (idx == blockDim.x - 1) {
    sh_din[idx + 3] = in[i+1];
    sh_din[idx + 4] = in[i+2];
  }
  __syncthreads();

  out[i] = (sh_din[idx] + sh_din[idx + 1] + sh_din[idx + 2] + sh_din[idx + 3] + sh_din[idx + 4]) / 5.0f;
}

__global__ void bar_tile_2(float out[], float in[])
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int idx = threadIdx.x;
  extern __shared__ float sh_din[];
  sh_din[idx] = in[i];
  __syncthreads();
  if (idx == 0) {
    out[i] = (in[i - 2] + in[i - 1] + sh_din[idx] + sh_din[idx + 1] + sh_din[idx + 2]) / 5.0f;
  }
  else if (idx == 1) {
    out[i] = (in[i - 2] + sh_din[idx - 1] + sh_din[idx] + sh_din[idx + 1] + sh_din[idx + 2]) / 5.0f;
  }
  else if (idx == blockDim.x - 2) {
    out[i] = (sh_din[idx - 2] + sh_din[idx - 1] + sh_din[idx] + sh_din[idx + 1] + in[i + 2]) / 5.0f;
  }
  else if (idx == blockDim.x - 1) {
    out[i] = (sh_din[idx - 2] + sh_din[idx - 1] + sh_din[idx] + in[i + 1] + in[i + 2]) / 5.0f;
  }
  else {
    out[i] = (sh_din[idx - 2] + sh_din[idx - 1] + sh_din[idx] + sh_din[idx + 1] + sh_din[idx + 2]) / 5.0f;
  }  
}

__global__ void bar_tile_3(float out[], float in[])
{
  int idx = threadIdx.x;
  extern __shared__ float sh_din[];
  int i_in = blockIdx.x * BLOCKSIZE + idx;
  sh_din[idx] = in[i_in-2];
  __syncthreads();
  if (idx < blockDim.x-4)
    out[i_in] = (sh_din[idx] + sh_din[idx + 1] + sh_din[idx + 2] + sh_din[idx + 3] + sh_din[idx + 4]) / 5.0f;
}

void cpuFoo(float out[], float A[], float B[], float C[], float D[], float E[])
{
	for (int i=0; i<N; i++)
	{
		out[i] = (A[i] + B[i] + C[i] + D[i] + E[i]) / 5.0f;
	}
}

void cpuBar(float out[], float in[])
{
	// ignore the boundaries
	for (int i=2; i<N-2; i++)
	{
		out[i] = (in[i-2] + in[i-1] + in[i] + in[i+1] + in[i+2]) / 5.0f;
	}
}

int main(int argc, char **argv)
{
	// declare and fill input arrays for foo() and bar()
	float fooA[N], fooB[N], fooC[N], fooD[N], fooE[N], barIn[N];
	for (int i=0; i<N; i++) 
	{
		fooA[i] = i; 
		fooB[i] = i+1;
		fooC[i] = i+2;
		fooD[i] = i+3;
		fooE[i] = i+4;
		barIn[i] = 2*i; 
	}
	// device arrays
	int numBytes = N * sizeof(float);
	float *d_fooA;	 	checkCudaErrors(hipMalloc(&d_fooA, numBytes));
	float *d_fooB; 		checkCudaErrors(hipMalloc(&d_fooB, numBytes));
	float *d_fooC;	 	checkCudaErrors(hipMalloc(&d_fooC, numBytes));
	float *d_fooD; 		checkCudaErrors(hipMalloc(&d_fooD, numBytes));
	float *d_fooE; 		checkCudaErrors(hipMalloc(&d_fooE, numBytes));
	float *d_barIn; 	checkCudaErrors(hipMalloc(&d_barIn, numBytes));
	checkCudaErrors(hipMemcpy(d_fooA, fooA, numBytes, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fooB, fooB, numBytes, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fooC, fooC, numBytes, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fooD, fooD, numBytes, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_fooE, fooE, numBytes, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_barIn, barIn, numBytes, hipMemcpyHostToDevice));	

	// output arrays for host and device
	float fooOut[N], barOut[N], *d_fooOut, *d_barOut;
	checkCudaErrors(hipMalloc(&d_fooOut, numBytes));
	checkCudaErrors(hipMalloc(&d_barOut, numBytes));

	// declare and compute reference solutions
	float ref_fooOut[N], ref_barOut[N]; 
	cpuFoo(ref_fooOut, fooA, fooB, fooC, fooD, fooE);
	cpuBar(ref_barOut, barIn);

	// launch and time foo and bar
	GpuTimer fooTimer, barTimer;
	fooTimer.Start();
	foo<<<N/BLOCKSIZE, BLOCKSIZE>>>(d_fooOut, d_fooA, d_fooB, d_fooC, d_fooD, d_fooE);
	fooTimer.Stop();
	
  checkCudaErrors(hipMemcpy(fooOut, d_fooOut, numBytes, hipMemcpyDeviceToHost));
  printf("foo<<<>>>(): %g ms elapsed. Verifying solution...", fooTimer.Elapsed());
  compareArrays(ref_fooOut, fooOut, N);

	barTimer.Start();
	bar<<<N/BLOCKSIZE, BLOCKSIZE>>>(d_barOut, d_barIn);
  //bar_tile << <N / BLOCKSIZE, BLOCKSIZE, (BLOCKSIZE + 4) * sizeof(float) >> >(d_barOut, d_barIn);
  //bar_tile_2 << <N / BLOCKSIZE, BLOCKSIZE, (BLOCKSIZE) * sizeof(float) >> >(d_barOut, d_barIn);
  //bar_tile_3 << <N / BLOCKSIZE, BLOCKSIZE+4, (BLOCKSIZE + 4) * sizeof(float) >> >(d_barOut, d_barIn);
	barTimer.Stop();

	checkCudaErrors(hipMemcpy(barOut, d_barOut, numBytes, hipMemcpyDeviceToHost));
	printf("bar<<<>>>(): %g ms elapsed. Verifying solution...", barTimer.Elapsed());
	compareArrays(ref_barOut, barOut, N);

  barTimer.Start();
  bar_tile << <N / BLOCKSIZE, BLOCKSIZE, (BLOCKSIZE + 4) * sizeof(float) >> >(d_barOut, d_barIn);
  barTimer.Stop();

  checkCudaErrors(hipMemcpy(barOut, d_barOut, numBytes, hipMemcpyDeviceToHost));
  printf("bar_tile<<<>>>(): %g ms elapsed. Verifying solution...", barTimer.Elapsed());
  compareArrays(ref_barOut, barOut, N);

  barTimer.Start();
  bar_tile_2 << <N / BLOCKSIZE, BLOCKSIZE, (BLOCKSIZE)* sizeof(float) >> >(d_barOut, d_barIn);
  barTimer.Stop();

  checkCudaErrors(hipMemcpy(barOut, d_barOut, numBytes, hipMemcpyDeviceToHost));
  printf("bar_tile_2<<<>>>(): %g ms elapsed. Verifying solution...", barTimer.Elapsed());
  compareArrays(ref_barOut, barOut, N);

  barTimer.Start();
  bar_tile_3 << <N / BLOCKSIZE, BLOCKSIZE + 4, (BLOCKSIZE + 4) * sizeof(float) >> >(d_barOut, d_barIn);
  barTimer.Stop();

  checkCudaErrors(hipMemcpy(barOut, d_barOut, numBytes, hipMemcpyDeviceToHost));
  printf("bar_tile_3<<<>>>(): %g ms elapsed. Verifying solution...", barTimer.Elapsed());
  compareArrays(ref_barOut, barOut, N);

	checkCudaErrors(hipFree(d_fooA));
	checkCudaErrors(hipFree(d_fooB));
	checkCudaErrors(hipFree(d_fooC));
	checkCudaErrors(hipFree(d_fooD));
	checkCudaErrors(hipFree(d_fooE));
  checkCudaErrors(hipFree(d_barIn));
	checkCudaErrors(hipFree(d_fooOut));
	checkCudaErrors(hipFree(d_barOut));
}
