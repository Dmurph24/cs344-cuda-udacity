#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <algorithm>
#include <hip/hip_runtime.h>

__global__ void shmem_reduce_kernel(float * d_out, const float * const d_in)
{
  // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
  extern __shared__ float sdata[];

  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int tid = threadIdx.x;

  // load shared mem from global mem
  sdata[tid] = d_in[myId];
  __syncthreads();            // make sure entire block is loaded!

  // do reduction in shared mem
  unsigned int s = blockDim.x / 2;
  // step 1: 分两半
  if (tid < s) {
    float temp = sdata[tid];
    sdata[tid] = max(temp, sdata[tid + s]);
    sdata[tid + s] = min(temp, sdata[tid + s]);
  }
  __syncthreads();        // make sure all adds at one stage are done!

  // step 2: 两边走
  for (s = s / 2; s > 0; s >>= 1)
  {
    if (tid < s) {
      sdata[tid] = max(sdata[tid], sdata[tid + s]);
    }
    else if (tid >= blockDim.x / 2 && tid < blockDim.x / 2 + s) {
      sdata[tid] = min(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();        // make sure all adds at one stage are done!
  }

  // only thread 0 writes result for this block back to global mem
  if (tid == 0)
  {
    d_out[blockIdx.x] = sdata[0];
    d_out[blockDim.x + blockIdx.x] = sdata[blockDim.x / 2];
    //printf("%f %f\n", sdata[0], sdata[blockDim.x / 2]); 
  }
}

__global__ void shmem_reduce_finish_kernel(float *min_logLum, 
  float *max_logLum, const float * const d_in)
{
  // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
  extern __shared__ float sdata[];
  int tid = threadIdx.x;

  // load shared mem from global mem
  sdata[tid] = d_in[tid];
  sdata[tid + blockDim.x] = d_in[tid + blockDim.x];
  __syncthreads();            // make sure entire block is loaded!

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s) {
      sdata[tid] = max(sdata[tid], sdata[tid + s]);
      sdata[tid + blockDim.x] = min(sdata[tid + blockDim.x], sdata[tid + blockDim.x + s]);
    }
    __syncthreads();        // make sure all adds at one stage are done!
  }

  // only thread 0 writes result for this block back to global mem
  if (tid == 0)
  {
    *max_logLum = sdata[0];
    *min_logLum = sdata[blockDim.x];
  }
}

void reduce(float *min_logLum, float *max_logLum, const float* const d_logLuminance, int length)
{
  // use reduce
  const int m = 1 << 6;
  int blocks = ceil((float)length / m);
  float *d_intermediate; // should not modify d_in
  hipMalloc(&d_intermediate, sizeof(float)* blocks * 2); // store max and min
  shmem_reduce_kernel<<<blocks, m, m * sizeof(float)>>>(d_intermediate, d_logLuminance);
  shmem_reduce_finish_kernel<<<1, blocks, 2 * blocks*sizeof(float)>>>(min_logLum, max_logLum, d_intermediate);
  hipFree(d_intermediate);
}

int main(int argc, char **argv)
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    hipSetDevice(dev);

    hipDeviceProp_t devProps;
    if (hipGetDeviceProperties(&devProps, dev) == 0)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %dB; compute v%d.%d; clock: %d kHz\n",
               devProps.name, (int)devProps.totalGlobalMem, 
               (int)devProps.major, (int)devProps.minor, 
               (int)devProps.clockRate);
    }

    const int ARRAY_SIZE = 1 << 12;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    float sum = 0.0f;
    for(int i = 0; i < ARRAY_SIZE; i++) {
        // generate random float in [-1.0f, 1.0f]
        h_in[i] = -1.0f + (float)rand()/((float)RAND_MAX/2.0f);
        sum += h_in[i];
    }

    // declare GPU memory pointers
    float *d_in;

    // allocate GPU memory
    hipMalloc((void **) &d_in, ARRAY_BYTES);

    // transfer the input array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // launch the kernel
    printf("Running reduce\n");
    float *d_min, *d_max;
    hipMalloc((void **) &d_min, sizeof(float));
    hipMalloc((void **) &d_max, sizeof(float));
    reduce(d_min, d_max, d_in, ARRAY_SIZE);

    // copy back the sum from GPU
    float h_min, h_max;
    hipMemcpy(&h_min, d_min, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(&h_max, d_max, sizeof(float), hipMemcpyDeviceToHost);

    printf("Max_GPU: %f  Min_GPU: %f\n", h_max, h_min);
    h_max = h_in[0]; h_min = h_in[0];
    for (size_t i = 1; i < ARRAY_SIZE; ++i) {
        h_max = std::max(h_in[i], h_max);
        h_min = std::min(h_in[i], h_min);
    }
    printf("Max_CPU: %f  Min_CPU: %f\n", h_max, h_min);

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_min);
    hipFree(d_max);
    return 0;
}
